// Using CUDA device to calculate pi
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NBIN  10000000  // Number of bins
#define NUM_BLOCK   32  // Number of thread blocks
#define NUM_THREAD 192  // Number of threads per block
int tid;
float pi = 0.0f;

// Kernel that executes on the CUDA device
__global__ void cal_pi(float *sum, int nbin, float step, int nthreads, int nblocks) {
	int i;
	float x;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
	for (i=idx; i< nbin; i+=nthreads*nblocks) {
		x = (i+0.5f)*step;
		sum[idx] += 4.0f/(1.0f+x*x);
	}
}

// Main routine that executes on the host
int main(void) {
	dim3 dimGrid(NUM_BLOCK,1,1);  // Grid dimensions
	dim3 dimBlock(NUM_THREAD,1,1);  // Block dimensions
	float *sumHost, *sumDev;  // Pointer to host & device arrays

	float step = 1.0f/NBIN;  // Step size
	size_t size = NUM_BLOCK*NUM_THREAD*sizeof(float);  //Array memory size
	sumHost = (float *)malloc(size);  //  Allocate array on host
	hipMalloc((void **) &sumDev, size);  // Allocate array on device
	// Initialize array in device to 0
	hipMemset(sumDev, 0, size);
	// Do calculation on device
	cal_pi <<<dimGrid, dimBlock>>> (sumDev, NBIN, step, NUM_THREAD, NUM_BLOCK); // call CUDA kernel
	// Retrieve result from device and store it in host array
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
	for(tid=0; tid<NUM_THREAD*NUM_BLOCK; tid++)
		pi += sumHost[tid];
	pi *= step;

	// Print results
	printf("PI = %f\n",pi);

	// Cleanup
	free(sumHost); 
	hipFree(sumDev);

	return 0;
}